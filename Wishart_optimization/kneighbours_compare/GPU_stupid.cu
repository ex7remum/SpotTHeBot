#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define BLOCK_DIM 16

//source https://github.com/vincentfpgarcia/kNN-CUDA

/**
 * Computes the squared Euclidean distance matrix between the query points and the reference points.
 *
 * @param ref          refence points stored in the global memory
 * @param ref_width    number of reference points
 * @param ref_pitch    pitch of the reference points array in number of column
 * @param query        query points stored in the global memory
 * @param query_width  number of query points
 * @param query_pitch  pitch of the query points array in number of columns
 * @param height       dimension of points = height of texture `ref` and of the array `query`
 * @param dist         array containing the query_width x ref_width computed distances
 */
__global__ void compute_distances(float * ref,
                                  int     ref_width,
                                  int     ref_pitch,
                                  float * query,
                                  int     query_width,
                                  int     query_pitch,
                                  int     height,
                                  float * dist) {

    // Declaration of the shared memory arrays As and Bs used to store the sub-matrix of A and B
    __shared__ float shared_A[BLOCK_DIM][BLOCK_DIM];
    __shared__ float shared_B[BLOCK_DIM][BLOCK_DIM];

    // Sub-matrix of A (begin, step, end) and Sub-matrix of B (begin, step)
    __shared__ int begin_A;
    __shared__ int begin_B;
    __shared__ int step_A;
    __shared__ int step_B;
    __shared__ int end_A;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Initialization of the SSD for the current thread
    float ssd = 0.f;

    // Loop parameters
    begin_A = BLOCK_DIM * blockIdx.y;
    begin_B = BLOCK_DIM * blockIdx.x;
    step_A  = BLOCK_DIM * ref_pitch;
    step_B  = BLOCK_DIM * query_pitch;
    end_A   = begin_A + (height-1) * ref_pitch;

    // Conditions
    int cond0 = (begin_A + tx < ref_width); // used to write in shared memory
    int cond1 = (begin_B + tx < query_width); // used to write in shared memory & to computations and to write in output array 
    int cond2 = (begin_A + ty < ref_width); // used to computations and to write in output matrix

    // Loop over all the sub-matrices of A and B required to compute the block sub-matrix
    for (int a = begin_A, b = begin_B; a <= end_A; a += step_A, b += step_B) {

        // Load the matrices from device memory to shared memory; each thread loads one element of each matrix
        if (a/ref_pitch + ty < height) {
            shared_A[ty][tx] = (cond0)? ref[a + ref_pitch * ty + tx] : 0;
            shared_B[ty][tx] = (cond1)? query[b + query_pitch * ty + tx] : 0;
        }
        else {
            shared_A[ty][tx] = 0;
            shared_B[ty][tx] = 0;
        }

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Compute the difference between the two matrixes; each thread computes one element of the block sub-matrix
        if (cond2 && cond1) {
            for (int k = 0; k < BLOCK_DIM; ++k){
                float tmp = shared_A[k][ty] - shared_B[k][tx];
                ssd += tmp*tmp;
            }
        }

        // Synchronize to make sure that the preceeding computation is done before loading two new sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory; each thread writes one element
    if (cond2 && cond1) {
        dist[ (begin_A + ty) * query_pitch + begin_B + tx ] = ssd;
    }
}

/**
 * For each reference point (i.e. each column) finds the k-th smallest distances
 * of the distance matrix and their respective indexes and gathers them at the top
 * of the 2 arrays.
 *
 * Since we only need to locate the k smallest distances, sorting the entire array
 * would not be very efficient if k is relatively small. Instead, we perform a
 * simple insertion sort by eventually inserting a given distance in the first
 * k values.
 *
 * @param dist         distance matrix
 * @param dist_pitch   pitch of the distance matrix given in number of columns
 * @param index        index matrix
 * @param index_pitch  pitch of the index matrix given in number of columns
 * @param width        width of the distance matrix and of the index matrix
 * @param height       height of the distance matrix
 * @param k            number of values to find
 */
__global__ void modified_insertion_sort(float * dist,
                                        int     dist_pitch,
                                        int *   index,
                                        int     index_pitch,
                                        int     width,
                                        int     height,
                                        int     k){

    // Column position
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;

    // Do nothing if we are out of bounds
    if (xIndex < width) {

        // Pointer shift
        float * p_dist  = dist  + xIndex;
        int *   p_index = index + xIndex;

        // Initialise the first index
        p_index[0] = 0;

        // Go through all points
        for (int i=1; i<height; ++i) {

            // Store current distance and associated index
            float curr_dist = p_dist[i*dist_pitch];
            int   curr_index  = i;

            // Skip the current value if its index is >= k and if it's higher the k-th slready sorted mallest value
            if (i >= k && curr_dist >= p_dist[(k-1)*dist_pitch]) {
                continue;
            }

            // Shift values (and indexes) higher that the current distance to the right
            int j = min(i, k-1);
            while (j > 0 && p_dist[(j-1)*dist_pitch] > curr_dist) {
                p_dist[j*dist_pitch]   = p_dist[(j-1)*dist_pitch];
                p_index[j*index_pitch] = p_index[(j-1)*index_pitch];
                --j;
            }

            // Write the current distance and index at their position
            p_dist[j*dist_pitch]   = curr_dist;
            p_index[j*index_pitch] = curr_index; 
        }
    }
}


/**
 * Computes the square root of the first k lines of the distance matrix.
 *
 * @param dist   distance matrix
 * @param width  width of the distance matrix
 * @param pitch  pitch of the distance matrix given in number of columns
 * @param k      number of values to consider
 */
__global__ void compute_sqrt(float * dist, int width, int pitch, int k){
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
    if (xIndex<width && yIndex<k)
        dist[yIndex*pitch + xIndex] = sqrt(dist[yIndex*pitch + xIndex]);
}

bool knn_cuda_global(const float * ref,
                     int           ref_nb,
                     const float * query,
                     int           query_nb,
                     int           dim,
                     int           k,
                     float *       knn_dist,
                     int *         knn_index) {

    // Constants
    const unsigned int size_of_float = sizeof(float);
    const unsigned int size_of_int   = sizeof(int);

    // Return variables
    hipError_t err0, err1, err2, err3;

    // Check that we have at least one CUDA device 
    int nb_devices;
    err0 = hipGetDeviceCount(&nb_devices);
    if (err0 != hipSuccess || nb_devices == 0) {
        printf("ERROR: No CUDA device found\n");
        return false;
    }

    // Select the first CUDA device as default
    err0 = hipSetDevice(0);
    if (err0 != hipSuccess) {
        printf("ERROR: Cannot set the chosen CUDA device\n");
        return false;
    }

    // Allocate global memory
    float * ref_dev   = NULL;
    float * query_dev = NULL;
    float * dist_dev  = NULL;
    int   * index_dev = NULL;
    size_t  ref_pitch_in_bytes;
    size_t  query_pitch_in_bytes;
    size_t  dist_pitch_in_bytes;
    size_t  index_pitch_in_bytes;
    err0 = hipMallocPitch((void**)&ref_dev,   &ref_pitch_in_bytes,   ref_nb   * size_of_float, dim);
    err1 = hipMallocPitch((void**)&query_dev, &query_pitch_in_bytes, query_nb * size_of_float, dim);
    err2 = hipMallocPitch((void**)&dist_dev,  &dist_pitch_in_bytes,  query_nb * size_of_float, ref_nb);
    err3 = hipMallocPitch((void**)&index_dev, &index_pitch_in_bytes, query_nb * size_of_int,   k);
    if (err0 != hipSuccess || err1 != hipSuccess || err2 != hipSuccess || err3 != hipSuccess) {
        printf("ERROR: Memory allocation error\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev); 
        return false;
    }

    // Deduce pitch values
    size_t ref_pitch   = ref_pitch_in_bytes   / size_of_float;
    size_t query_pitch = query_pitch_in_bytes / size_of_float;
    size_t dist_pitch  = dist_pitch_in_bytes  / size_of_float;
    size_t index_pitch = index_pitch_in_bytes / size_of_int;

    // Check pitch values
    if (query_pitch != dist_pitch || query_pitch != index_pitch) {
        printf("ERROR: Invalid pitch value\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev); 
        return false; 
    }

    // Copy reference and query data from the host to the device
    err0 = hipMemcpy2D(ref_dev,   ref_pitch_in_bytes,   ref,   ref_nb * size_of_float,   ref_nb * size_of_float,   dim, hipMemcpyHostToDevice);
    err1 = hipMemcpy2D(query_dev, query_pitch_in_bytes, query, query_nb * size_of_float, query_nb * size_of_float, dim, hipMemcpyHostToDevice);
    if (err0 != hipSuccess || err1 != hipSuccess) {
        printf("ERROR: Unable to copy data from host to device\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev); 
        return false; 
    }

    // Compute the squared Euclidean distances
    dim3 block0(BLOCK_DIM, BLOCK_DIM, 1);
    dim3 grid0(query_nb / BLOCK_DIM, ref_nb / BLOCK_DIM, 1);
    if (query_nb % BLOCK_DIM != 0) grid0.x += 1;
    if (ref_nb   % BLOCK_DIM != 0) grid0.y += 1;
    compute_distances<<<grid0, block0>>>(ref_dev, ref_nb, ref_pitch, query_dev, query_nb, query_pitch, dim, dist_dev);
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev); 
        return false;
    }

    // Sort the distances with their respective indexes
    dim3 block1(256, 1, 1);
    dim3 grid1(query_nb / 256, 1, 1);
    if (query_nb % 256 != 0) grid1.x += 1;
    modified_insertion_sort<<<grid1, block1>>>(dist_dev, dist_pitch, index_dev, index_pitch, query_nb, ref_nb, k);
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev); 
        return false;
    }

    // Compute the square root of the k smallest distances
    dim3 block2(16, 16, 1);
    dim3 grid2(query_nb / 16, k / 16, 1);
    if (query_nb % 16 != 0) grid2.x += 1;
    if (k % 16 != 0)        grid2.y += 1;
    compute_sqrt<<<grid2, block2>>>(dist_dev, query_nb, query_pitch, k);	
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev); 
        return false;
    }

    // Copy k smallest distances / indexes from the device to the host
    err0 = hipMemcpy2D(knn_dist,  query_nb * size_of_float, dist_dev,  dist_pitch_in_bytes,  query_nb * size_of_float, k, hipMemcpyDeviceToHost);
    err1 = hipMemcpy2D(knn_index, query_nb * size_of_int,   index_dev, index_pitch_in_bytes, query_nb * size_of_int,   k, hipMemcpyDeviceToHost);
    if (err0 != hipSuccess || err1 != hipSuccess) {
        printf("ERROR: Unable to copy data from device to host\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev); 
        return false; 
    }

    // Memory clean-up
    hipFree(ref_dev);
    hipFree(query_dev);
    hipFree(dist_dev);
    hipFree(index_dev); 

    return true;
}

#include <algorithm>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <time.h>

/**
 * Initializes randomly the reference and query points.
 *
 * @param ref        refence points
 * @param ref_nb     number of reference points
 * @param query      query points
 * @param query_nb   number of query points
 * @param dim        dimension of points
 */
void initialize_data(float * ref,
                     int     ref_nb,
                     float * query,
                     int     query_nb,
                     int     dim) {

    // Get reference points
     srand(time(NULL));
    for (int i=0; i<ref_nb*dim; ++i) {
        ref[i] = 10. * (float)(rand() / (double)RAND_MAX);
    }

    // Get query points
    for (int i=0; i<query_nb*dim; ++i) {
       query[i] = ref[i];
    }
}


/**
 * Test an input k-NN function implementation by verifying that its output
 * results (distances and corresponding indexes) are similar to the expected
 * results (ground truth).
 *
 * Since the k-NN computation might end-up in slightly different results
 * compared to the expected one depending on the considered implementation,
 * the verification consists in making sure that the accuracy is high enough.
 *
 * The tested function is ran several times in order to have a better estimate
 * of the processing time.
 *
 * @param ref            reference points
 * @param ref_nb         number of reference points
 * @param query          query points
 * @param query_nb       number of query points
 * @param dim            dimension of reference and query points
 * @param k              number of neighbors to consider
 * @param gt_knn_dist    ground truth distances
 * @param gt_knn_index   ground truth indexes
 * @param knn            function to test
 * @param name           name of the function to test (for display purpose)
 * @param nb_iterations  number of iterations
 * return false in case of problem, true otherwise
 */
bool test(const float * ref,
          int           ref_nb,
          const float * query,
          int           query_nb,
          int           dim,
          int           k,
          float *       gt_knn_dist,
          int *         gt_knn_index,
          bool (*knn)(const float *, int, const float *, int, int, int, float *, int *),
          const char *  name,
          int           nb_iterations) {

    // Allocate memory for computed k-NN neighbors
    float * test_knn_dist  = (float*) malloc(query_nb * k * sizeof(float));
    int   * test_knn_index = (int*)   malloc(query_nb * k * sizeof(int));

    // Allocation check
    if (!test_knn_dist || !test_knn_index) {
        printf("ALLOCATION ERROR\n");
        free(test_knn_dist);
        free(test_knn_index);
        return false;
    }

    // Start timer
    struct timeval tic;
    gettimeofday(&tic, NULL);

    // Compute k-NN several times
    for (int i=0; i<nb_iterations; ++i) {
        if (!knn(ref, ref_nb, query, query_nb, dim, k, test_knn_dist, test_knn_index)) {
            free(test_knn_dist);
            free(test_knn_index);
            return false;
        }
    }

    // Stop timer
    struct timeval toc;
    gettimeofday(&toc, NULL);

    // Elapsed time in ms
    double elapsed_time = toc.tv_sec - tic.tv_sec;
    elapsed_time += (toc.tv_usec - tic.tv_usec) / 1000000.;

    for (int i = 0; i < query_nb * k; ++i) {
        printf("%d ", test_knn_index[i]);
    }
    for (int i = 0; i < query_nb * k; ++i) {
        printf("%f ", test_knn_dist[i]);
    }

    printf("\nPASSED in %8.5f seconds (averaged over %3d iterations)\n", elapsed_time / nb_iterations, nb_iterations);
    free(test_knn_dist);
    free(test_knn_index);

    return true;
}

int main(void) {

    // Parameters
    const int ref_nb   = 1000;
    const int query_nb = 1000;
    const int dim      = 4;
    const int k        = 11;

    // Allocate input points and output k-NN distances / indexes
    float * ref        = (float*) malloc(ref_nb   * dim * sizeof(float));
    float * query      = (float*) malloc(query_nb * dim * sizeof(float));
    float * knn_dist   = (float*) malloc(query_nb * k   * sizeof(float));
    int   * knn_index  = (int*)   malloc(query_nb * k   * sizeof(int));

    // Allocation checks
    if (!ref || !query || !knn_dist || !knn_index) {
        printf("Error: Memory allocation error\n"); 
        free(ref);
	    free(query);
	    free(knn_dist);
	    free(knn_index);
        return EXIT_FAILURE;
    }

    // Initialize reference and query points with random values
    initialize_data(ref, ref_nb, query, query_nb, dim);


    // Test all k-NN functions
    test(ref, ref_nb, query, query_nb, dim, k, knn_dist, knn_index, &knn_cuda_global,  "knn_cuda_global",  1); 
    // Deallocate memory 
    free(ref);
    free(query);
    free(knn_dist);
    free(knn_index);

    return EXIT_SUCCESS;
}
